#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void multmat(float *dst, float *src1, float *src2, int n){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	if(i<n && j<n){
		float sum = 0;
		for(int k = 0; k<n; k++)
			sum += src1[i*n+k]*src2[k*n+j];
		dst[i*n+j] = sum;
	}
}

int main(){
	int n = 1000;
	float *A = (float*)malloc(n*n*sizeof(float));
	float *B = (float*)malloc(n*n*sizeof(float));
	float *C = (float*)malloc(n*n*sizeof(float));
	float *dA, *dB, *dC;
	
	//memory space allocation on the GPU
	hipMalloc(&dA, n*n*sizeof(float));
	hipMalloc(&dB, n*n*sizeof(float));
	hipMalloc(&dC, n*n*sizeof(float));

	for(int i = 0; i<n; i++){
		for(int j = 0; j<n; j++){
			int idx = (i*n + j);
			A[idx] = idx;
			B[idx] = n*n-idx;
		}
	}

	//transfer the host matrix to the GPU memory
	hipMemcpy(dA, A, n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, n*n*sizeof(float), hipMemcpyHostToDevice);	
	
	dim3 blockSize(8,8);
	dim3 nBlocks(n/blockSize.x + 1, n/blockSize.y + 1);
	multmat<<<nBlocks,blockSize>>>(dC,dA,dB,n);

	//transfer the matrix back to the host memory
	hipMemcpy(C, dC, n*n*sizeof(float), hipMemcpyDeviceToHost);
	/*
	
	for(int i = 0; i<n; i++){
		for(int j = 0; j<n; j++)
			printf("%.2f ", A[i*n + j]);
		printf("\b\n");
	}
	printf("\n");


	for(int i = 0; i<n; i++){
		for(int j = 0; j<n; j++)
			printf("%.2f ", B[i*n + j]);
		printf("\b\n");
	}
	printf("\n");


	for(int i = 0; i<n; i++){
		for(int j = 0; j<n; j++)
			printf("%.2f ", C[i*n + j]);
		printf("\b\n");
	}
	printf("\n");
	*/
	free(A);
	free(B);
	free(C);
	//free the GPU memory space
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
}


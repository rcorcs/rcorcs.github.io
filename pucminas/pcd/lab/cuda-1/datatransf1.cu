#include <stdio.h>

#include <hip/hip_runtime.h>

int main(){
        int n = 100;
        int *vec = (int*)malloc(n*sizeof(int));
        int *dvec;
        //memory space allocation on the GPU
        hipMalloc(&dvec, n*sizeof(int));
        for(int i = 0; i<n; i++)
                vec[i] = i;

        //transfer the host vector to the GPU memory
        hipMemcpy(dvec, vec, n*sizeof(int), hipMemcpyHostToDevice);
        for(int i = 0; i<n; i++)
                vec[i] = 0;

        //transfer the vector back to the host memory
        hipMemcpy(vec, dvec, n*sizeof(int), hipMemcpyDeviceToHost);

        for(int i = 0; i<n; i++)
                printf("%d ", vec[i]);
        printf("\b\n");

	//zero the vector on the GPU memory space
	hipMemset(dvec, 0, n*sizeof(n));

        //transfer the vector back to the host memory
        hipMemcpy(vec, dvec, n*sizeof(int), hipMemcpyDeviceToHost);

        for(int i = 0; i<n; i++)
                printf("%d ", vec[i]);
        printf("\b\n");

        free(vec);
        //free the GPU memory space
        hipFree(dvec);
}

